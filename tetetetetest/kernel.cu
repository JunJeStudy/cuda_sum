#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


//#define array_size 100000000
#define array_size 100

//987459712


hipError_t addWithCuda(int *c, unsigned int size);

__device__ int temp[array_size];

__global__ void addKernel(int *c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	temp[tid] += tid + 1;
	int i_sum = tid;

	if (i_sum % 2 == 0){ i_sum /= 2; }
	else { i_sum /= 2 + 1; }

	for (int i = i_sum; i > 0;){
		temp[tid] += temp[i];
		i_sum /= 2;
		__syncthreads();
	}

	c[tid] = temp[tid];
}
int main()
{
	int a[array_size];
	memset(a, 0, array_size * sizeof(int));
	int c[array_size] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, array_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (int i = 0; i < array_size; i++){
		printf("%d - %d\n", i, c[i]);
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, unsigned int size)
{
	int *dev_a = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <10, 10 >> >(dev_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);

	return cudaStatus;
}
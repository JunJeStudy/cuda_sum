#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


//#define array_size 100000000
#define array_size 101

//987459712


hipError_t addWithCuda(int *total);

__shared__ int temp[array_size];

__global__ void addKernel(int *tid_c, int *tid_total)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	tid_c[tid] = tid;
	if (tid <= array_size)
	{
		
			temp[threadIdx.x] = tid;

			if (threadIdx.x==0)
			{
				for(int i=0;i<=blockDim.x;i++)
				{
					//__syncthreads();
					atomicAdd(tid_total, temp[i]);
					//__syncthreads();
					//printf("i = %d \n", *tid_total);
				}
			}

	}

}
int main() 
{

	int	total=0;
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(&total);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
/*
	for (int i = 0; i < array_size; i++){
		printf("%d - %d\n", i, c[i]);
	}*/
	printf("%d", total);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *total)
{
	int *a;
	int *cuda_total;
	//int *dev_a = 0;
	//int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc(&a, sizeof(int)*array_size);
	cudaStatus = hipMalloc(&cuda_total, sizeof(int)); 
	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(32);
	dim3 dimGrid((array_size + dimBlock.x - 1) / dimBlock.x);

	addKernel << <dimGrid, dimBlock >> >(a,cuda_total);

		// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	printf("tetst %d\n", *total);
	cudaStatus = hipMemcpy(total, cuda_total, sizeof(int), hipMemcpyDeviceToHost);
	printf("tetst %d\n", *total);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	

Error:
	hipFree(a);
	hipFree(cuda_total);

	return cudaStatus;
}